#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
This source distribution is placed in the public domain by its author,
Jason Papadopoulos. You may use it for any purpose, free of charge,
without having to notify anyone. I disclaim any responsibility for any
errors.

Optionally, please be nice and tell me if you find this source to be
useful. Again optionally, if you add to the functionality present here
please consider making those additions public too, so that others may 
benefit from your work.	

$Id$
--------------------------------------------------------------------*/

#include "stage1_core.h"

#ifdef __cplusplus
extern "C" {
#endif

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_trans_32(uint32 *p_array, uint32 num_p, uint32 *start_roots,
			uint32 num_roots, uint32 *p_out, uint32 *roots_out,
			specialq_t *q_batch, uint32 num_specialq, 
			uint32 specialq_block, uint32 num_entries, uint32 shift)
{
	uint32 p, pp, pp_w, p_offset;
	uint32 specialq_start, specialq_end;
	uint32 q, qq_prod, qq_prod_offset, curr_offset, q_count;
	uint32 i, j, k, m, start_i, gcd, inv, curr_inv;
	uint32 qroot, newroot;
	specialq_t *curr_q;

	p_offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (p_offset >= num_p)
		return;

	p = p_array[p_offset];
	pp = p * p;
	pp_w = montmul32_w(pp);

	specialq_start = blockIdx.y * specialq_block;
	specialq_end = __min(specialq_start + specialq_block, num_specialq);

	qq_prod_offset = specialq_start * num_entries + p_offset;
	curr_q = q_batch + specialq_start;

	q = j = 0;
	for (i = specialq_start; j == 0 && i < specialq_end; i++) {

		if (q != curr_q->p) {
			q = curr_q->p;
			gcd = gcd32(p, q);
			if (gcd == 1)
				j = qq_prod = curr_q->pp % pp;
		}

		roots_out[qq_prod_offset] = j;
		qq_prod_offset += num_entries;
		curr_q++;
	}
	if (j == 0)
		return;

	for (start_i = i - 1; i < specialq_end; i++) {

		if (q != curr_q->p) {
			q = curr_q->p;
			gcd = gcd32(p, q);

			if (gcd == 1)
				j = qq_prod = montmul32(qq_prod, 
						curr_q->pp % pp, 
						pp, pp_w);
			else
				j = 0;
		}

		roots_out[qq_prod_offset] = j;
		qq_prod_offset += num_entries;
		curr_q++;
	}

	inv = modinv32(qq_prod, pp);
	inv = montmul32(inv, montmul32_r(pp), pp, pp_w);
	qq_prod_offset -= num_entries;

	for (i--; i > start_i; i--) {

		uint32 curr_qq_prod = roots_out[qq_prod_offset];

		if (curr_qq_prod > 0)
			break;

		qq_prod_offset -= num_entries;
	}

	curr_offset = qq_prod_offset - num_entries;
	q = i;
	q_count = 1;

	for (i--; (int32)i >= (int32)start_i; 
			i--, curr_offset -= num_entries) {

		uint32 curr_qq_prod = roots_out[curr_offset];

		if (curr_qq_prod == 0) {
			continue;
		}
		else if (curr_qq_prod == qq_prod) {
			q_count++;
			continue;
		}

		curr_inv = montmul32(curr_qq_prod, inv, pp, pp_w);
		inv = montmul32(inv, q_batch[q].pp % pp, pp, pp_w);

		do {
			qroot = q_batch[q].root % pp;

			for (j = qq_prod_offset, k = p_offset, m = 0; 
						m < num_roots; 
						j += num_p, k += num_p, m++) {

				newroot = modsub32(start_roots[k], 
							qroot, pp);
				newroot = montmul32(newroot, curr_inv, 
							pp, pp_w);
				if (newroot > pp / 2)
					newroot -= pp;

				p_out[j] = (q << shift) | p;
				roots_out[j] = newroot;
			}

			q--;
			qq_prod_offset -= num_entries;
		} while (--q_count);

		q = i;
		q_count = 1;
		qq_prod = curr_qq_prod;
		qq_prod_offset = curr_offset;
	}

	curr_inv = inv;
	while ((int32)q >= (int32)start_i) {

		qroot = q_batch[q].root % pp;

		for (j = qq_prod_offset, k = p_offset, m = 0; 
					m < num_roots; 
					j += num_p, k += num_p, m++) {

			newroot = modsub32(start_roots[k], 
						qroot, pp);
			newroot = montmul32(newroot, curr_inv, 
						pp, pp_w);
			if (newroot > pp / 2)
				newroot -= pp;

			p_out[j] = (q << shift) | p;
			roots_out[j] = newroot;
		}

		q--;
		qq_prod_offset -= num_entries;
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_trans_64(uint32 *p_array, uint32 num_p, uint64 *start_roots,
			uint32 num_roots, uint32 *p_out, int64 *roots_out,
			specialq_t *q_batch, uint32 num_specialq, 
			uint32 specialq_block, uint32 num_entries, uint32 shift)
{
	uint32 offset, i, j, p, pp_w, q, end, gcd;
	uint64 pp, pp_r, qq, tmp, inv, newroot;
	uint32 specialq_start, specialq_end;

	offset = blockIdx.x * blockDim.x + threadIdx.x;
	if (offset >= num_p)
		return;

	p = p_array[offset];
	pp = wide_sqr32(p);
	pp_w = montmul32_w((uint32)pp);
	pp_r = montmul64_r(pp, pp_w);
	end = num_p * num_roots;

	specialq_start = blockIdx.y * specialq_block;
	specialq_end = __min(specialq_start + specialq_block, num_specialq);

	q = 0;
	for (i = specialq_start; i < specialq_end; i++) {
		if (q != q_batch[i].p) {
			q = q_batch[i].p;
			gcd = gcd32(p, q);

			if (gcd == 1) {
				qq = wide_sqr32(q) % pp;
				tmp = modinv32(q % p, p);
				tmp = wide_sqr32(tmp);
				tmp = montmul64(tmp, pp_r, pp, pp_w);
				inv = montmul64(qq, tmp, pp, pp_w);
				inv = modsub64((uint64)2, inv, pp);
				inv = montmul64(inv, tmp, pp, pp_w);
				inv = montmul64(inv, pp_r, pp, pp_w);
			}
		}

		for (j = offset; j < end; j += num_p) {

			if (gcd == 1) {
				newroot = modsub64(start_roots[j],
						q_batch[i].root % pp, pp);
				newroot = montmul64(newroot, inv, pp, pp_w);

				if (newroot > pp / 2)
					newroot -= pp;

				p_out[j + num_entries * i] = (i << shift) | p;
				roots_out[j + num_entries * i] = newroot;
			}
		}
	}
}

/*------------------------------------------------------------------------*/
__device__ void
store_hit(found_t *found_array, uint32 found_array_size,
		uint32 p1, uint32 p2,
		int64 root, specialq_t *q)
{
	/* don't use atomicInc because we don't want
	   wraparound to occur */

	uint32 index = atomicAdd(&found_array[0].p1, 1);

	if (index < found_array_size - 1) {

		found_t *f = found_array + index + 1;

		f->p1 = p1;
		f->p2 = p2;
		f->q = q->p;
		f->qroot = q->root;
		f->offset = root;
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_final_32(uint32 *p_array, int32 *roots, uint32 num_entries,
			specialq_t * q_batch, uint32 num_specialq, 
			found_t *found_array, uint32 shift)
{
	uint32 i, j;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32 mask = (1 << shift) - 1;
	uint32 p_array_size = num_entries * num_specialq;

	for (i = my_threadid; i < p_array_size - 1; i += num_threads) {

		int32 root1 = roots[i];
		uint32 p1 = p_array[i];

		if (root1 == 0)
			continue;

		for (j = i + 1; j < p_array_size; j++) {
			int32 root2 = roots[j];
			uint32 p2 = p_array[j];

			if (root1 != root2)
				break;

			if (p1 >= p2 &&
			    (p1 >> shift) == (p2 >> shift) &&
			    gcd32( (p1 & mask), (p2 & mask) ) == 1) {

				store_hit(found_array, FOUND_ARRAY_SIZE,
						p1 & mask, p2 & mask, 
						(int64)root1,
						q_batch + (p1 >> shift));
			}
		}

		for (j = i - 1; (int32)j >= 0; j--) {
			int64 root2 = roots[j];
			uint32 p2 = p_array[j];

			if (root1 != root2)
				break;

			if (p1 >= p2 &&
			    (p1 >> shift) == (p2 >> shift) &&
			    gcd32( (p1 & mask), (p2 & mask) ) == 1) {

				store_hit(found_array, FOUND_ARRAY_SIZE,
						p1 & mask, p2 & mask, 
						(int64)root1,
						q_batch + (p1 >> shift));
			}
		}
	}
}

/*------------------------------------------------------------------------*/
__global__ void
sieve_kernel_final_64(uint32 *p_array, int64 *roots, uint32 num_entries,
			specialq_t * q_batch, uint32 num_specialq, 
			found_t *found_array, uint32 shift)
{
	uint32 i, j;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 my_threadid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32 mask = (1 << shift) - 1;
	uint32 p_array_size = num_entries * num_specialq;

	for (i = my_threadid; i < p_array_size - 1; i += num_threads) {

		int64 root1 = roots[i];
		uint32 p1 = p_array[i];

		if (root1 == 0)
			continue;

		for (j = i + 1; j < p_array_size; j++) {
			int64 root2 = roots[j];
			uint32 p2 = p_array[j];

			if (root1 != root2)
				break;

			if (p1 >= p2 &&
			    (p1 >> shift) == (p2 >> shift) &&
			    gcd32( (p1 & mask), (p2 & mask) ) == 1) {

				store_hit(found_array, FOUND_ARRAY_SIZE,
						p1 & mask, p2 & mask, root1,
						q_batch + (p1 >> shift));
			}
		}

		for (j = i - 1; (int32)j >= 0; j--) {
			int64 root2 = roots[j];
			uint32 p2 = p_array[j];

			if (root1 != root2)
				break;

			if (p1 >= p2 &&
			    (p1 >> shift) == (p2 >> shift) &&
			    gcd32( (p1 & mask), (p2 & mask) ) == 1) {

				store_hit(found_array, FOUND_ARRAY_SIZE,
						p1 & mask, p2 & mask, root1,
						q_batch + (p1 >> shift));
			}
		}
	}
}

#ifdef __cplusplus
}
#endif
