#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
This source distribution is placed in the public domain by its author,
Jason Papadopoulos. You may use it for any purpose, free of charge,
without having to notify anyone. I disclaim any responsibility for any
errors.

Optionally, please be nice and tell me if you find this source to be
useful. Again optionally, if you add to the functionality present here
please consider making those additions public too, so that others may 
benefit from your work.	

$Id$
--------------------------------------------------------------------*/

typedef int int32;
typedef unsigned int uint32;
typedef unsigned long long uint64;
typedef long long int64;

#ifdef __cplusplus
extern "C" {
#endif

/*------------------------------------------------------------------------*/
__global__ void
lanczos_kernel_mask(uint64 *x, uint64 mask, uint32 n)
{
	uint32 i;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 grid_id = blockIdx.x * blockDim.x + threadIdx.x;

	for (i = grid_id; i < n; i += num_threads)
		x[i] &= mask;
}

/*------------------------------------------------------------------------*/
__global__ void
lanczos_kernel_xor(uint64 *dest, uint64 *src, uint32 n)
{
	uint32 i;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 grid_id = blockIdx.x * blockDim.x + threadIdx.x;

	for (i = grid_id; i < n; i += num_threads)
		dest[i] ^= src[i];
}

/*------------------------------------------------------------------------*/
texture<uint2, hipTextureType1D, hipReadModeElementType> inner_tex;

__device__ uint64
uint2_to_uint64(uint2 v)
{
	return (uint64)v.y << 32 | v.x;
}

__device__ uint32
bfe(uint64 x, uint32 pos, uint32 bits)
{
#if __CUDA_ARCH__ >= 200

	uint32 res;
	uint32 hi = (uint32)(x >> 32);
	uint32 lo = (uint32)x;

	if (pos < 32) {
	       if (pos + bits > 32) {
			res = ((lo >> pos) | (hi << (32 - pos))) &
				((1 << bits) - 1);
	       }
	       else {
			asm("bfe.u32 %0, %1, %2, %3; \n\t"
				: "=r"(res) : "r"(lo), "r"(pos), "r"(bits));
	       }
	}
	else {
		asm("bfe.u32 %0, %1, %2, %3; \n\t"
			: "=r"(res) : "r"(hi), "r"(pos - 32), "r"(bits));
	}

	return res;

#else
	return (x >> pos) & ((1 << bits) - 1);
#endif
}

__global__ void
lanczos_kernel_inner_prod(uint64 *y, uint64 *v, uint32 n)
{
	uint32 i;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 grid_id = blockIdx.x * blockDim.x + threadIdx.x;

	for (i = grid_id; i < n; i += num_threads) {

		uint64 vi = v[i];
		uint64 yi = y[i];
		yi ^=  uint2_to_uint64(tex1Dfetch(inner_tex, 
					0*64 + (int)bfe(vi,  0, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					1*64 + (int)bfe(vi,  6, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					2*64 + (int)bfe(vi, 12, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					3*64 + (int)bfe(vi, 18, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					4*64 + (int)bfe(vi, 24, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					5*64 + (int)bfe(vi, 30, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					6*64 + (int)bfe(vi, 36, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					7*64 + (int)bfe(vi, 42, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					8*64 + (int)bfe(vi, 48, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					9*64 + (int)bfe(vi, 54, 6)))
		       ^ uint2_to_uint64(tex1Dfetch(inner_tex, 
					10*64 + (int)bfe(vi, 60, 6)));
		y[i] = yi;
	}
}

/*------------------------------------------------------------------------*/
/* thanks to Patrick Stach for ideas on this */

#define MAX_OUTER_THREADS 256

__global__ void
lanczos_kernel_outer_prod(uint64 *x, uint64 *y,
			uint32 *xy, uint32 n)
{
	uint32 i;
	uint32 num_threads = gridDim.x * blockDim.x;
	uint32 grid_id = blockIdx.x * blockDim.x + threadIdx.x;
	uint32 block_id = threadIdx.x;
	__shared__ uint64 scratch[3 * MAX_OUTER_THREADS];
	uint64 *s = scratch + (block_id & ~0x1f);

	scratch[block_id + 0*MAX_OUTER_THREADS] = 0;
	scratch[block_id + 1*MAX_OUTER_THREADS] = 0;
	scratch[block_id + 2*MAX_OUTER_THREADS] = 0;

	for (i = grid_id; i < n; i += num_threads) {

		uint32 j; 
		uint32 k = block_id & 0x1f;
		uint64 xi = x[i];
		uint64 yi = y[i];

		if (k != 0)
			xi = (xi >> (2 * k)) | (xi << (64 - (2 * k)));

#pragma unroll
		for (j = 0; j < 32; j++) {

			uint32 off = bfe(xi, 2 * j, 2);
			uint64 tmp = yi;

			if (off == 0) {
				tmp = 0;
				off = 1;
			}

			s[((k + j) & 0x1f) + 
				MAX_OUTER_THREADS * (off - 1)] ^= tmp;
		}
	}

	s = scratch + block_id;
	__syncthreads();
	s[0*MAX_OUTER_THREADS] ^= s[2*MAX_OUTER_THREADS];
	s[1*MAX_OUTER_THREADS] ^= s[2*MAX_OUTER_THREADS];
	__syncthreads();

	for (i = MAX_OUTER_THREADS / 2; i >= 32; i >>= 1) {
		if (block_id < i) {
			s[0*MAX_OUTER_THREADS] ^= s[0*MAX_OUTER_THREADS + i];
			s[1*MAX_OUTER_THREADS] ^= s[1*MAX_OUTER_THREADS + i];
		}
		__syncthreads();
	}


	if (block_id < 64) {
		uint32 *t = (uint32 *)scratch;

		i = 4 * (block_id / 2);

		if (block_id % 2 == 0)
			atomicXor(&xy[i], t[block_id]);
		else
			atomicXor(&xy[i + 1], t[block_id]);
	}
	else if (block_id < 128) {
		uint32 *t = (uint32 *)(scratch + MAX_OUTER_THREADS);

		i = 4 * ((block_id - 64) / 2) + 2;

		if (block_id % 2 == 0)
			atomicXor(&xy[i], t[block_id - 64]);
		else
			atomicXor(&xy[i + 1], t[block_id - 64]);
	}
}

#ifdef __cplusplus
}
#endif
